#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// compile with: nvcc hello.cu -o hello
// run with: ./hello

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    return 0;
}
